#include "hip/hip_runtime.h"
#include <cstdlib>
#include <sys/time.h>
#include <math.h>
#include "wave.h"

__global__ void calcularFrame(unsigned char* pic, int width)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int frame = 0; frame < frames; frame++) {
        for (int row = index; row < width; row += stride) {
          for (int col = 0; col < width; col++) {
            float fx = col - 1024/2;
            float fy = row - 1024/2;
            float d = sqrtf( fx * fx + fy * fy );
            unsigned char color = (unsigned char) (160.0f + 127.0f *
                                              cos(d/10.0f - frame/7.0f) /
                                              (d/50.0f + 1.0f));
            pic[frame * width * width + row * width + col] = (unsigned char) color;
          }
        }
    }
    
}

int main(int argc, char *argv[])
{

    // check command line
    if (argc != 3) {fprintf(stderr, "usage: %s frame_width num_frames\n", argv[0]); exit(-1);}
    int width = atoi(argv[1]);
    if (width < 100) {fprintf(stderr, "error: frame_width must be at least 100\n"); exit(-1);}
    int frames = atoi(argv[2]);
    if (frames < 1) {fprintf(stderr, "error: num_frames must be at least 1\n"); exit(-1);}
    printf("computing %d frames of %d by %d picture\n", frames, width, width);

     // allocate picture array
    unsigned char* pic;

    // start time
    timeval start, end;
    gettimeofday(&start, NULL);

    hipMallocManaged(&pic, frames * width * width * sizeof(char));

    int blockSize = 256;
    int numBlocks = (width + blockSize - 1) / blockSize;

    calcularFrame<<<numBlocks,blockSize>>>(pic,width,frames);

    // verify result by writing frames to BMP files
    if ((width <= 256) && (frames <= 100)) {
      for (int frame = 0; frame < frames; frame++) {
        char name[32];
        sprintf(name, "wave%d.bmp", frame + 1000);
        writeBMP(width, width, &pic[frame * width * width], name);
      }
    }

    hipFree(pic);

    // end time
    gettimeofday(&end, NULL);
    double runtime = end.tv_sec + end.tv_usec / 1000000.0 - start.tv_sec - start.tv_usec / 1000000.0;
    printf("compute time: %.4f s\n", runtime);

    return 0;
}
